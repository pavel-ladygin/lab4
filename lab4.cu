#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>

#define CUDA_CHECK(call) \
    do { hipError_t err = call; if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(1); } } while(0)

void computePowerCPU(const float* input, float* output, int size, float exponent) {
    for (int i = 0; i < size; ++i) {
        output[i] = powf(input[i], exponent);
    }
}

__global__ void computePowerGPU(const float* input, float* output, int size, float exponent) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = powf(input[idx], exponent);
    }
}

void rotateImageCPU(const unsigned char* src, unsigned char* dst, int w, int h) {
    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < w; ++x) {
            int newIdx = y * w + x;
            int oldIdx = (h - x - 1) * w + y;
            dst[newIdx] = src[oldIdx];
        }
    }
}

__global__ void rotateImageGPU(const unsigned char* src, unsigned char* dst, int w, int h) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h) {
        int newIdx = y * w + x;
        int oldIdx = (h - x - 1) * w + y;
        dst[newIdx] = src[oldIdx];
    }
}

int main() {
    srand(42);

    // Task 1: Power Array
    const int n = 500000;
    const float p = 0.5f;
    float *dataIn, *resultCpu, *resultGpu;
    float *devIn, *devOut;

    dataIn = (float*)malloc(n * sizeof(float));
    resultCpu = (float*)malloc(n * sizeof(float));
    resultGpu = (float*)malloc(n * sizeof(float));

    for (int i = 0; i < n; ++i) {
        dataIn[i] = (float)(rand() % 1000) / 100.0f;
    }
    memset(resultCpu, 0, n * sizeof(float));
    memset(resultGpu, 0, n * sizeof(float));

    CUDA_CHECK(hipMalloc(&devIn, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&devOut, n * sizeof(float)));
    CUDA_CHECK(hipMemset(devOut, 0, n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(devIn, dataIn, n * sizeof(float), hipMemcpyHostToDevice));

    auto start = std::chrono::high_resolution_clock::now();
    computePowerCPU(dataIn, resultCpu, n, p);
    auto end = std::chrono::high_resolution_clock::now();
    double cpuTime = std::chrono::duration<double, std::milli>(end - start).count();

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    hipEvent_t startEvent, stopEvent;
    CUDA_CHECK(hipEventCreate(&startEvent));
    CUDA_CHECK(hipEventCreate(&stopEvent));
    CUDA_CHECK(hipEventRecord(startEvent));
    computePowerGPU<<<gridSize, blockSize>>>(devIn, devOut, n, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(stopEvent));
    CUDA_CHECK(hipEventSynchronize(stopEvent));
    float gpuTime;
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, startEvent, stopEvent));

    CUDA_CHECK(hipMemcpy(resultGpu, devOut, n * sizeof(float), hipMemcpyDeviceToHost));

    bool task1Match = true;
    for (int i = 0; i < n; ++i) {
        if (fabs(resultCpu[i] - resultGpu[i]) > 1e-5) {
            task1Match = false;
            break;
        }
    }

    printf("Task 1: Power Array\nCPU Time: %.2f ms\nGPU Time: %.2f ms\nResults match: %s\nSample: A[0]=%.2f, B_cpu[0]=%.2f, B_gpu[0]=%.2f\n",
           cpuTime, gpuTime, task1Match ? "Yes" : "No", dataIn[0], resultCpu[0], resultGpu[0]);

    free(dataIn);
    free(resultCpu);
    free(resultGpu);
    CUDA_CHECK(hipFree(devIn));
    CUDA_CHECK(hipFree(devOut));
    CUDA_CHECK(hipEventDestroy(startEvent));
    CUDA_CHECK(hipEventDestroy(stopEvent));

    // Task 2: Rotate Image
    const int w = 512;
    const int h = 512;
    unsigned char *imgIn, *imgOutCpu, *imgOutGpu;
    unsigned char *devImgIn, *devImgOut;

    imgIn = (unsigned char*)malloc(w * h * sizeof(unsigned char));
    imgOutCpu = (unsigned char*)malloc(w * h * sizeof(unsigned char));
    imgOutGpu = (unsigned char*)malloc(w * h * sizeof(unsigned char));

    for (int i = 0; i < w * h; ++i) {
        imgIn[i] = (unsigned char)(rand() % 256);
    }
    memset(imgOutCpu, 0, w * h * sizeof(unsigned char));
    memset(imgOutGpu, 0, w * h * sizeof(unsigned char));

    CUDA_CHECK(hipMalloc(&devImgIn, w * h * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&devImgOut, w * h * sizeof(unsigned char)));
    CUDA_CHECK(hipMemset(devImgOut, 0, w * h * sizeof(unsigned char)));
    CUDA_CHECK(hipMemcpy(devImgIn, imgIn, w * h * sizeof(unsigned char), hipMemcpyHostToDevice));

    start = std::chrono::high_resolution_clock::now();
    rotateImageCPU(imgIn, imgOutCpu, w, h);
    end = std::chrono::high_resolution_clock::now();
    cpuTime = std::chrono::duration<double, std::milli>(end - start).count();

    dim3 blockDim(16, 16);
    dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
    CUDA_CHECK(hipEventCreate(&startEvent));
    CUDA_CHECK(hipEventCreate(&stopEvent));
    CUDA_CHECK(hipEventRecord(startEvent));
    rotateImageGPU<<<gridDim, blockDim>>>(devImgIn, devImgOut, w, h);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(stopEvent));
    CUDA_CHECK(hipEventSynchronize(stopEvent));
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, startEvent, stopEvent));

    CUDA_CHECK(hipMemcpy(imgOutGpu, devImgOut, w * h * sizeof(unsigned char), hipMemcpyDeviceToHost));

    bool task2Match = true;
    for (int i = 0; i < w * h; ++i) {
        if (imgOutCpu[i] != imgOutGpu[i]) {
            task2Match = false;
            break;
        }
    }

    printf("\nTask 2: Rotate Image\nCPU Time: %.2f ms\nGPU Time: %.2f ms\nResults match: %s\nSample: input[0]=%u, output_cpu[0]=%u, output_gpu[0]=%u\n",
           cpuTime, gpuTime, task2Match ? "Yes" : "No", imgIn[0], imgOutCpu[0], imgOutGpu[0]);

    free(imgIn);
    free(imgOutCpu);
    free(imgOutGpu);
    CUDA_CHECK(hipFree(devImgIn));
    CUDA_CHECK(hipFree(devImgOut));
    CUDA_CHECK(hipEventDestroy(startEvent));
    CUDA_CHECK(hipEventDestroy(stopEvent));

    return 0;
}
